#include "hip/hip_runtime.h"
#include "Updater.h"


Updater::Updater(void)
{
}


Updater::~Updater(void)
{
}

__global__ void addKernel(std::vector<VertexCell> & cellList1, int xDimension)
{
	int neighbourCount = 0;
	//32 x 16 blocks
	// 24 x 48 grid of 32x16 blocks = 768 x 768
	/*int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (cellList1[(i * xDimension) + j].color == 1){

					if (cellList1[((i * xDimension) + (j - 1))].color == 0){ //left						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + (j - 1))].color == 0){ //top left						
						neighbourCount++;					
					}
					if (cellList1[(((i + 1) * xDimension) + (j - 1))].color == 0){ //bottom left					
						neighbourCount++;						
					}
					if (cellList1[((i * xDimension) + (j + 1))].color == 0){ //right						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + (j + 1))].color == 0){ //top right						
						neighbourCount++;						
					}
					if (cellList1[(((i + 1) * xDimension) + (j + 1))].color == 0){ //bottom right						
						neighbourCount++;						
					}
					if (cellList1[(((i + 1) * xDimension) + j)].color == 0){ //bottom						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + j)].color == 0){ //top						
						neighbourCount++;						
					}
					if (neighbourCount >= 6){						
						cellList1[(i * xDimension) + j].color = 0;
						cellList1[(i * xDimension) + j].r = 1.0f;
						cellList1[(i * xDimension) + j].g = 0.0f;
						cellList1[(i * xDimension) + j].b = 0.0f;
					}
				}else if (cellList1[(i * xDimension) + j].color == 0){

					if (cellList1[((i * xDimension) + (j - 1))].color == 2){ //left						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + (j - 1))].color == 2){ //top left						
						neighbourCount++;					
					}
					if (cellList1[(((i + 1) * xDimension) + (j - 1))].color == 2){ //bottom left					
						neighbourCount++;						
					}
					if (cellList1[((i * xDimension) + (j + 1))].color == 2){ //right						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + (j + 1))].color == 2){ //top right						
						neighbourCount++;						
					}
					if (cellList1[(((i + 1) * xDimension) + (j + 1))].color == 2){ //bottom right						
						neighbourCount++;						
					}
					if (cellList1[(((i + 1) * xDimension) + j)].color == 2){ //bottom						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + j)].color == 2){ //top						
						neighbourCount++;						
					}
					if (neighbourCount >= 6){						
						cellList1[(i * xDimension) + j].color = 1;
						cellList1[(i * xDimension) + j].r = 0.0f;
						cellList1[(i * xDimension) + j].g = 1.0f;
						cellList1[(i * xDimension) + j].b = 0.0f;
					}
				}			
				neighbourCount = 0;		
			*/
}

void changeVertexColor(VertexCell &vc, float r, float g, float b){
	vc.r = r;
	vc.g = g;
	vc.b = b;
}

/*struct updateCancerStruct {

	std::vector<VertexCell> &cellList1;
	int xDimension;
	int yDimension;
	mutable int neighbourCount;

	void operator()(const tbb::blocked_range2d<size_t>& r) const  {
			
		for (size_t i = r.rows().begin(); i != r.rows().end(); ++i){
			for (size_t j = r.cols().begin(); j != r.cols().end(); ++j) {
				if (cellList1[(i * xDimension) + j].color == 1){

					if (cellList1[((i * xDimension) + (j - 1))].color == 0){ //left						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + (j - 1))].color == 0){ //top left						
						neighbourCount++;					
					}
					if (cellList1[(((i + 1) * xDimension) + (j - 1))].color == 0){ //bottom left					
						neighbourCount++;						
					}
					if (cellList1[((i * xDimension) + (j + 1))].color == 0){ //right						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + (j + 1))].color == 0){ //top right						
						neighbourCount++;						
					}
					if (cellList1[(((i + 1) * xDimension) + (j + 1))].color == 0){ //bottom right						
						neighbourCount++;						
					}
					if (cellList1[(((i + 1) * xDimension) + j)].color == 0){ //bottom						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + j)].color == 0){ //top						
						neighbourCount++;						
					}
					if (neighbourCount >= 6){						
						cellList1[(i * xDimension) + j].color = 0;
						cellList1[(i * xDimension) + j].r = 1.0f;
						cellList1[(i * xDimension) + j].g = 0.0f;
						cellList1[(i * xDimension) + j].b = 0.0f;
					}
				}else if (cellList1[(i * xDimension) + j].color == 0){

					if (cellList1[((i * xDimension) + (j - 1))].color == 2){ //left						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + (j - 1))].color == 2){ //top left						
						neighbourCount++;					
					}
					if (cellList1[(((i + 1) * xDimension) + (j - 1))].color == 2){ //bottom left					
						neighbourCount++;						
					}
					if (cellList1[((i * xDimension) + (j + 1))].color == 2){ //right						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + (j + 1))].color == 2){ //top right						
						neighbourCount++;						
					}
					if (cellList1[(((i + 1) * xDimension) + (j + 1))].color == 2){ //bottom right						
						neighbourCount++;						
					}
					if (cellList1[(((i + 1) * xDimension) + j)].color == 2){ //bottom						
						neighbourCount++;						
					}
					if (cellList1[(((i - 1) * xDimension) + j)].color == 2){ //top						
						neighbourCount++;						
					}
					if (neighbourCount >= 6){						
						cellList1[(i * xDimension) + j].color = 1;
						cellList1[(i * xDimension) + j].r = 0.0f;
						cellList1[(i * xDimension) + j].g = 1.0f;
						cellList1[(i * xDimension) + j].b = 0.0f;
					}
				}			
				neighbourCount = 0;		
			}		
		}
	}
	updateCancerStruct(std::vector<VertexCell> & cellList, int xDim, int yDim) : cellList1(cellList), xDimension(xDim), yDimension(yDim) {}
};*/

void Updater::updateCancer(std::vector<VertexCell> & cellList1, int xDimension, int yDimension, int size){
	
	/*tbb::parallel_for(tbb::blocked_range2d<size_t>(1, (xDimension-1), 1000, 1, (yDimension-1), 1000),
		updateCancerStruct(cellList1, xDimension, yDimension));*/
	int N = 10;
	int *dev_data = 0;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_data, cellList1.size() * sizeof(VertexCell));
	if (cudaStatus != hipSuccess) { 
		fprintf(stderr, "hipMalloc failed!");
		exit(1);
	}

	cudaStatus = hipMemcpy(dev_data, &cellList1[0], size * sizeof(VertexCell), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
	 fprintf(stderr, "hipMemcpy failed!");
	 exit(1);
	}

	addKernel<<<N, size>>>(cellList1, 768);

}